
#include <hip/hip_runtime.h>
/*
 * Simple kernel for ptxjit demonstration.
 *
 */ 
 extern "C" __global__ void myKernel(int *data)
 {
 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	data[tid] = tid;
 }
