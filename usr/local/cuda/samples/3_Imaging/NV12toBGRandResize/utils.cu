#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2019 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdlib.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "resize_convert.h"
#include "utils.h"

__global__ void floatToChar(float *src, unsigned char *dst, int height,
                            int width, int batchSize) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;

  if (x >= height * width) return;

  int offset = height * width * 3;

  for (int j = 0; j < batchSize; j++) {
    // b
    *(dst + j * offset + x * 3 + 0) =
        (unsigned char)*(src + j * offset + height * width * 0 + x);
    // g
    *(dst + j * offset + x * 3 + 1) =
        (unsigned char)*(src + j * offset + height * width * 1 + x);
    // r
    *(dst + j * offset + x * 3 + 2) =
        (unsigned char)*(src + j * offset + height * width * 2 + x);
  }
}

void floatPlanarToChar(float *src, unsigned char *dst, int height, int width,
                       int batchSize) {
  floatToChar<<<(height * width - 1) / 1024 + 1, 1024, 0, NULL>>>(
      src, dst, height, width, batchSize);
}

void dumpRawBGR(float *d_srcBGR, int pitch, int width, int height,
                int batchSize, char *folder, char *tag) {
  float *bgr, *d_bgr;
  int frameSize;
  char directory[120];
  char mkdir_cmd[256];
#if !defined(_WIN32)
  sprintf(directory, "output/%s", folder);
  sprintf(mkdir_cmd, "mkdir -p %s 2> /dev/null", directory);
#else
  sprintf(directory, "output\\%s", folder);
  sprintf(mkdir_cmd, "mkdir %s 2> nul", directory);
#endif

  int ret = system(mkdir_cmd);

  frameSize = width * height * 3 * sizeof(float);
  bgr = (float *)malloc(frameSize);
  if (bgr == NULL) {
    std::cerr << "Failed malloc for bgr\n";
    return;
  }

  d_bgr = d_srcBGR;
  for (int i = 0; i < batchSize; i++) {
    char filename[120];
    std::ofstream *outputFile;

    checkCudaErrors(hipMemcpy((void *)bgr, (void *)d_bgr, frameSize,
                               hipMemcpyDeviceToHost));
    sprintf(filename, "%s/%s_%d.raw", directory, tag, (i + 1));

    outputFile = new std::ofstream(filename);
    if (outputFile) {
      outputFile->write((char *)bgr, frameSize);
      delete outputFile;
    }

    d_bgr += pitch * height * 3;
  }

  free(bgr);
}

void dumpBGR(float *d_srcBGR, int pitch, int width, int height, int batchSize,
             char *folder, char *tag) {
  dumpRawBGR(d_srcBGR, pitch, width, height, batchSize, folder, tag);
}

void dumpYUV(unsigned char *d_nv12, int size, char *folder, char *tag) {
  unsigned char *nv12Data;
  std::ofstream *nv12File;
  char filename[120];
  char directory[60];
  char mkdir_cmd[256];
#if !defined(_WIN32)
  sprintf(directory, "output/%s", folder);
  sprintf(mkdir_cmd, "mkdir -p %s 2> /dev/null", directory);
#else
  sprintf(directory, "output\\%s", folder);
  sprintf(mkdir_cmd, "mkdir %s 2> nul", directory);
#endif

  int ret = system(mkdir_cmd);

  sprintf(filename, "%s/%s.nv12", directory, tag);

  nv12File = new std::ofstream(filename);
  if (nv12File == NULL) {
    std::cerr << "Failed to new " << filename;
    return;
  }

  nv12Data = (unsigned char *)malloc(size * (sizeof(char)));
  if (nv12Data == NULL) {
    std::cerr << "Failed to allcoate memory\n";
    return;
  }

  hipMemcpy((void *)nv12Data, (void *)d_nv12, size, hipMemcpyDeviceToHost);

  nv12File->write((const char *)nv12Data, size);

  free(nv12Data);
  delete nv12File;
}
