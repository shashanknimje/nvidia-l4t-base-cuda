#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2019 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Implements BGR 3 progressive planars frames batch resize

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "resize_convert.h"

__global__ void resizeBGRplanarBatchKernel(hipTextureObject_t texSrc,
    float *pDst, int nDstPitch, int nDstHeight, int nSrcHeight,
    int batch, float scaleX, float scaleY,
    int cropX, int cropY, int cropW, int cropH) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= (int)(cropW/scaleX) || y >= (int)(cropH/scaleY))
        return;

    int frameSize = nDstPitch*nDstHeight;
    float *p = NULL;
    for (int i = blockIdx.z; i < batch; i += gridDim.z) {
        #pragma unroll
        for (int channel=0; channel < 3; channel++){
            p = pDst + i * 3 * frameSize + y * nDstPitch + x + channel * frameSize;
            *p = tex2D<float>(texSrc, x * scaleX + cropX,
                                ((3 * i + channel) * nSrcHeight + y * scaleY + cropY));
        }
    }
}


static void resizeBGRplanarBatchCore(
        float *dpSrc, int nSrcPitch, int nSrcWidth, int nSrcHeight,
        float *dpDst, int nDstPitch, int nDstWidth, int nDstHeight,
        int nBatchSize, hipStream_t stream, bool whSameResizeRatio,
        int cropX, int cropY, int cropW, int cropH) {
    hipTextureObject_t texSrc[2];
    int nTiles = 1, h, iTile;

    h = nSrcHeight * 3 * nBatchSize;
    while ((h + nTiles - 1) / nTiles > 65536)
        nTiles++;

    if (nTiles > 2)
        return;

    int batchTile = nBatchSize / nTiles;
    int batchTileLast = nBatchSize - batchTile * (nTiles-1);

    for (iTile = 0; iTile < nTiles; ++iTile) {
        int bs = (iTile == nTiles - 1) ? batchTileLast : batchTile;
        float *dpSrcNew = dpSrc +
            iTile * (batchTile * 3 * nSrcHeight * nSrcPitch);

        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypePitch2D;
        resDesc.res.pitch2D.devPtr = dpSrcNew;
        resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
        resDesc.res.pitch2D.width = nSrcWidth;
        resDesc.res.pitch2D.height = bs * 3 * nSrcHeight;
        resDesc.res.pitch2D.pitchInBytes = nSrcPitch * sizeof(float);
        hipTextureDesc texDesc = {};
        texDesc.filterMode = hipFilterModeLinear;
        texDesc.readMode = hipReadModeElementType;

        hipCreateTextureObject(&texSrc[iTile], &resDesc, &texDesc, NULL);
        float *dpDstNew = dpDst +
            iTile * (batchTile * 3 * nDstHeight * nDstPitch);

        if(cropW == 0 || cropH == 0) {
            cropX = 0;
            cropY = 0;
            cropW = nSrcWidth;
            cropH = nSrcHeight;
        }

        float scaleX = (cropW*1.0f / nDstWidth);
        float scaleY = (cropH*1.0f / nDstHeight);

        if(whSameResizeRatio == true)
            scaleX = scaleY = scaleX > scaleY ? scaleX : scaleY;
        dim3 block(32, 32, 1);

        size_t blockDimZ = bs;
        // Restricting blocks in Z-dim till 32 to not launch too many blocks
        blockDimZ = (blockDimZ > 32) ? 32 : blockDimZ;
        dim3 grid((cropW*1.0f/scaleX + block.x - 1) / block.x,
                  (cropH*1.0f/scaleY + block.y - 1) / block.y, blockDimZ);

        resizeBGRplanarBatchKernel<<<grid, block, 0, stream>>>
                (texSrc[iTile], dpDstNew, nDstPitch, nDstHeight, nSrcHeight,
                bs, scaleX, scaleY, cropX, cropY, cropW, cropH);

    }

    for (iTile = 0; iTile < nTiles; ++iTile)
        hipDestroyTextureObject(texSrc[iTile]);
}

void resizeBGRplanarBatch(
        float *dpSrc, int nSrcPitch, int nSrcWidth, int nSrcHeight,
        float *dpDst, int nDstPitch, int nDstWidth, int nDstHeight,
        int nBatchSize, hipStream_t stream,
        int cropX, int cropY, int cropW, int cropH, bool whSameResizeRatio) {
    resizeBGRplanarBatchCore(dpSrc, nSrcPitch, nSrcWidth, nSrcHeight,
        dpDst, nDstPitch, nDstWidth, nDstHeight, nBatchSize, stream,
        whSameResizeRatio, cropX, cropY, cropW, cropH);
}
