#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2019 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Implements interlace NV12 frames batch resize

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "resize_convert.h"

__global__ static void resizeNV12BatchKernel(hipTextureObject_t texSrcLuma,
                                             hipTextureObject_t texSrcChroma,
                                             uint8_t *pDstNv12, int nSrcWidth,
                                             int nSrcHeight, int nDstPitch,
                                             int nDstWidth, int nDstHeight,
                                             int nBatchSize) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  int px = x * 2, py = y * 2;

  if ((px + 1) >= nDstWidth || (py + 1) >= nDstHeight) return;

  float fxScale = 1.0f * nSrcWidth / nDstWidth;
  float fyScale = 1.0f * nSrcHeight / nDstHeight;

  uint8_t *p = pDstNv12 + px + py * nDstPitch;
  int hh = nDstHeight * 3 / 2;
  int nByte = nDstPitch * hh;
  int px_fxScale = px * fxScale;
  int px_fxScale_1 = (px + 1) * fxScale;
  int py_fyScale = py * fyScale;
  int py_fyScale_1 = (py + 1) * fyScale;

  for (int i = blockIdx.z; i < nBatchSize; i+=gridDim.z) {
    *(uchar2 *)p = make_uchar2(tex2D<uint8_t>(texSrcLuma, px_fxScale, py_fyScale),
                          tex2D<uint8_t>(texSrcLuma, px_fxScale_1, py_fyScale));
    *(uchar2 *)(p + nDstPitch) =
        make_uchar2(tex2D<uint8_t>(texSrcLuma, px_fxScale, py_fyScale_1),
               tex2D<uint8_t>(texSrcLuma, px_fxScale_1, py_fyScale_1));
    *(uchar2 *)(p + (nDstHeight - y) * nDstPitch) = tex2D<uchar2>(
        texSrcChroma, x * fxScale, (hh * i + nDstHeight + y) * fyScale);
    p += nByte;
    py += hh;
  }
}

void resizeNV12Batch(uint8_t *dpSrc, int nSrcPitch, int nSrcWidth,
                     int nSrcHeight, uint8_t *dpDst, int nDstPitch,
                     int nDstWidth, int nDstHeight, int nBatchSize,
                     hipStream_t stream) {
  int hhSrc = ceilf(nSrcHeight * 3.0f / 2.0f);
  hipResourceDesc resDesc = {};
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = dpSrc;
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<uint8_t>();
  resDesc.res.pitch2D.width = nSrcWidth;
  resDesc.res.pitch2D.height = hhSrc * nBatchSize;
  resDesc.res.pitch2D.pitchInBytes = nSrcPitch;

  hipTextureDesc texDesc = {};
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;

  hipTextureObject_t texLuma = 0;
  checkCudaErrors(hipCreateTextureObject(&texLuma, &resDesc, &texDesc, NULL));

  resDesc.res.pitch2D.desc = hipCreateChannelDesc<uchar2>();
  resDesc.res.pitch2D.width /= 2;

  hipTextureObject_t texChroma = 0;
  checkCudaErrors(hipCreateTextureObject(&texChroma, &resDesc, &texDesc, NULL));

  dim3 block(32, 32, 1);

  size_t blockDimZ = nBatchSize;

  // Restricting blocks in Z-dim till 32 to not launch too many blocks
  blockDimZ = (blockDimZ > 32) ? 32 : blockDimZ;

  dim3 grid((nDstWidth / 2 + block.x) / block.x,
            (nDstHeight / 2 + block.y) / block.y, blockDimZ);
  resizeNV12BatchKernel<<<grid, block, 0, stream>>>(
      texLuma, texChroma, dpDst, nSrcWidth, nSrcHeight, nDstPitch, nDstWidth,
      nDstHeight, nBatchSize);

  checkCudaErrors(hipDestroyTextureObject(texLuma));
  checkCudaErrors(hipDestroyTextureObject(texChroma));
}
