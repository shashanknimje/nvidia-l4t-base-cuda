#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2018 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "commonKernels.hpp"

__global__ void spinWhileLessThanOne(volatile unsigned int *latch) {
  while (latch[0] < 1)
    ;
}
